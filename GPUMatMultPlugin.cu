#include "hip/hip_runtime.h"
#include "GPUMatMultPlugin.h"


void GPUMatMultPlugin::input(std::string infile) {
  readParameterFile(infile);
}

void GPUMatMultPlugin::run() {}

void GPUMatMultPlugin::output(std::string outfile) {
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA;
  float *deviceB;
  float *deviceC;
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;
  int numCColumns;
int M, N, P;
 M = atoi(myParameters["M"].c_str());
 N = atoi(myParameters["N"].c_str());
 P = atoi(myParameters["P"].c_str());
 numARows = M;
 numAColumns = N;
 numBRows = N;
 numBColumns = P;
 numCRows = M;
 numCColumns = P;

  hostA = (float*) malloc (M*N*sizeof(float));
  hostB = (float*) malloc (N*P*sizeof(float));
  //@@ Allocate the hostC matrix
  hostC = (float *)malloc(numARows * numBColumns * sizeof(float));

  numCRows    = numARows;
  numCColumns = numBColumns;

 std::ifstream myinput((std::string(PluginManager::prefix())+myParameters["matrix1"]).c_str(), std::ios::in);
 int i;
 for (i = 0; i < M*N; ++i) {
	float k;
	myinput >> k;
        hostA[i] = k;
 }
 std::ifstream myinput2((std::string(PluginManager::prefix())+myParameters["matrix2"]).c_str(), std::ios::in);
 for (i = 0; i < N*P; ++i) {
	float k;
	myinput2 >> k;
        hostB[i] = k;
 }

  //@@ Allocate GPU memory here
  hipMalloc((void **)&deviceA,
                     numARows * numAColumns * sizeof(float));
  hipMalloc((void **)&deviceB,
                     numBRows * numBColumns * sizeof(float));
  hipMalloc((void **)&deviceC,
                     numARows * numBColumns * sizeof(float));
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceA, hostA,
                     numARows * numAColumns * sizeof(float),
                     hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB,
                     numBRows * numBColumns * sizeof(float),
                     hipMemcpyHostToDevice);

  //@@ Initialize the grid and block dimensions here
  dim3 blockDim(16, 16);
// changed to BColumns and ARows from Acolumns and BRows
  dim3 gridDim(ceil(((float)numBColumns) / blockDim.x),
               ceil(((float)numARows) / blockDim.y));

  //@@ Launch the GPU Kernel here
  sgemm<<<gridDim, blockDim>>>(deviceA, deviceB, deviceC, numARows,
                               numAColumns, numBRows, numBColumns);
  hipDeviceSynchronize();
  hipMemcpy(hostC, deviceC,
                     numARows * numBColumns * sizeof(float),
                     hipMemcpyDeviceToHost);
	std::ofstream outsfile(outfile.c_str(), std::ios::out);

        for (i = 0; i < M*P; ++i){
		outsfile << hostC[i];//std::setprecision(0) << a[i*N+j];
		outsfile << "\n";
	}
  //@@ Free the GPU memory here
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);

  free(hostA);
  free(hostB);
  free(hostC);

}
PluginProxy<GPUMatMultPlugin> GPUMatMultPluginProxy = PluginProxy<GPUMatMultPlugin>("GPUMatMult", PluginManager::getInstance());


